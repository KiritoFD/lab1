#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

// CUDA核函数，并行计算多个序列的滚动哈希值
__global__ void rollingHashKernel(const char* sequences, int num_sequences, int sequence_length,
                                  int prime, int mod, const int* base_map, int* hash_values) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_sequences) {
        long long hash_val = 0;
        for (int i = 0; i < sequence_length; ++i) {
            hash_val = (hash_val * prime + base_map[sequences[idx * sequence_length + i]]) % mod;
        }
        hash_values[idx] = static_cast<int>(hash_val);
    }
}

// CPU函数，用于调用CUDA核函数
void calculateRollingHashes(const std::vector<std::string>& sequences, int prime, int mod,
                            const std::unordered_map<char, int>& base_map, std::vector<int>& hash_values) {
    int num_sequences = sequences.size();
    int sequence_length = sequences[0].length();

    // 将base_map转换为可以在CUDA中使用的int数组
    std::vector<int> cuda_base_map(256, 0);
    for (int i = 0; i < 256; ++i) {
        cuda_base_map[i] = base_map.count(static_cast<char>(i)) ? base_map.at(static_cast<char>(i)) : 0;
    }

    // 将序列数据复制到连续的内存区域
    std::vector<char> sequence_data;
    for (const auto& seq : sequences) {
        sequence_data.insert(sequence_data.end(), seq.begin(), seq.end());
    }

    // 在GPU上分配内存
    char* d_sequences;
    int* d_hash_values;
    int* d_base_map;
    hipMalloc(&d_sequences, num_sequences * sequence_length * sizeof(char));
    hipMalloc(&d_hash_values, num_sequences * sizeof(int));
    hipMalloc(&d_base_map, 256 * sizeof(int));

    // 将数据复制到GPU
    hipMemcpy(d_sequences, sequence_data.data(), num_sequences * sequence_length * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_hash_values, hash_values.data(), num_sequences * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_base_map, cuda_base_map.data(), 256 * sizeof(int), hipMemcpyHostToDevice);

    // 配置CUDA核函数
    int threads_per_block = 256;
    int num_blocks = (num_sequences + threads_per_block - 1) / threads_per_block;

    // 调用CUDA核函数
    rollingHashKernel<<<num_blocks, threads_per_block>>>(d_sequences, num_sequences, sequence_length, prime, mod, d_base_map, d_hash_values);

    // 将结果复制回CPU
    hipMemcpy(hash_values.data(), d_hash_values, num_sequences * sizeof(int), hipMemcpyDeviceToHost);

    // 释放GPU内存
    hipFree(d_sequences);
    hipFree(d_hash_values);
    hipFree(d_base_map);
}
